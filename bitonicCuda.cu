#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>

#include <hip/hip_runtime.h>


extern "C" void allocateMemory(int **arr, int arraySize)
{
     hipMallocManaged(arr, ( (arraySize* sizeof(int))));
}

extern "C" void callCudaFree(int* local)
{
	hipFree(local);
}

//extern void callMPI(int* local,int* arr,int arrSize,int mpi_size,int x_rank);


extern "C" void cudaInit( int myrank)
{
	int cE;
    int cudaDeviceCount = 1;

    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n",
        cE, cudaDeviceCount );
        exit(-1);
    }
    if( (cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess )
    {
        printf(" Unable to have rank %d set to cuda device %d, error is %d \n",
        myrank, (myrank % cudaDeviceCount), cE);
        exit(-1);
    }
}

__global__ void mergeKernel(int j, int mpi_size, int mpi_rank, int *arr, int arrSize, int sizeCompare,int* prev_local, int* next_local)
{	
	//nt *prev_local = NULL;
    //int *next_local = NULL;

	bool sameVal = false;
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int global_idx = i + arrSize / mpi_size * mpi_rank;
    int x = global_idx ^ j;
    int x_rank = x / (arrSize / mpi_size);
    if ( global_idx >= x ) {
        if ( mpi_rank == x_rank ) {
            if(sameVal == false)
            {
                sameVal = true;
            }
        } 
        else {
            if ( prev_local == NULL ) {
                //prev_local = calloc(arrSize / mpi_size, sizeof(int));
                //allocateMemory(&prev_local,arrSize/mpi_size);
                prev_local = arr + arrSize / mpi_size * x_rank;
                //callMPI(prev_local,arr,arrSize,mpi_size,x_rank);
            }

            if ( (sizeCompare & x) == 0 && arr[i] < prev_local[i] ) {
                arr[i] = prev_local[i];
            }
            if ( (sizeCompare & x) != 0 && arr[i] > prev_local[i] ) {
                arr[i] = prev_local[i];
            }
        }
    }
    else { 
        if ( x_rank == mpi_rank ) {
            int y = x - arrSize / mpi_size * mpi_rank;
            if ( (global_idx & sizeCompare) == 0 && arr[i] > arr[y] ) {
                int temp = arr[i];
                arr[i] = arr[y];
                arr[y] = temp;
            }
            if ( (global_idx & sizeCompare) != 0 && arr[i] < arr[y] ) {
                int temp = arr[i];
                arr[i] = arr[y];
                arr[y] = temp;
            }
        }
        else {
            if ( next_local == NULL ) {
                //next_local = calloc(arrSize / mpi_size, sizeof(int));
                //allocateMemory(&next_local,arrSize/mpi_size);
                next_local = arr + arrSize / mpi_size * x_rank;
                //callMPI(next_local,arr,arrSize,mpi_size,x_rank);
            }

            if ( (global_idx & sizeCompare) == 0 && arr[i] > next_local[i] ) {
                arr[i] = next_local[i];
            }
            if ( (global_idx & sizeCompare) != 0 && arr[i] < next_local[i] ) {
                arr[i] = next_local[i];
            }
        }
    }
}

extern "C" void mergeKernelLaunch(int blockSize,int threadsCount,int j, int mpi_size, int mpi_rank, int *arr, int arrSize, int sizeCompare,int* prev_local, int* next_local)
{
	mergeKernel<<<blockSize,threadsCount>>>(j, mpi_size, mpi_rank, arr, arrSize, sizeCompare, prev_local, next_local);
}